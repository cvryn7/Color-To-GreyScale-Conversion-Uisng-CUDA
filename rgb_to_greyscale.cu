#include "hip/hip_runtime.h"
/**
*Developed By Karan Bhagat
*February 2017
**/

#include <stdio.h>
#include <string>
#include <math.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

//number of channels i.e. R G B
#define CHANNELS 3

//Cuda kernel for converting RGB image into a GreyScale image
__global__
void colorConvertToGrey(unsigned char *rgb, unsigned char *grey, int rows, int cols)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	//Compute for only those threads which map directly to 
	//image grid
	if (col < cols && row < rows)
	{
		int grey_offset = row * cols + col;
		int rgb_offset = grey_offset * CHANNELS;
	
    	unsigned char r = rgb[rgb_offset + 0];
	    unsigned char g = rgb[rgb_offset + 1];
	    unsigned char b = rgb[rgb_offset + 2];
	
	    grey[grey_offset] = r * 0.299f + g * 0.587f + b * 0.114f;
    }
}

size_t loadImageFile(unsigned char *grey_image, const std::string &input_file, int *rows, int *cols );

void outputImage(const std::string &output_file, unsigned char *grey_image, int rows, int cols);

unsigned char *h_rgb_image; //store image's rbg data

int main(int argc, char **argv) 
{
	std::string input_file;
	std::string output_file;

	//Check for the input file and output file names
	switch(argc) {
		case 3:
			input_file = std::string(argv[1]);
			output_file = std::string(argv[2]);
            break;
		default:
			std::cerr << "Usage: <executable> input_file output_file";
			exit(1);
	}
	
	unsigned char *d_rgb_image; //array for storing rgb data on device
	unsigned char *h_grey_image, *d_grey_image; //host and device's grey data array pointers
	int rows; //number of rows of pixels
	int cols; //number of columns of pixels
	
	//load image into an array and retrieve number of pixels
	const size_t total_pixels = loadImageFile(h_grey_image, input_file, &rows, &cols);

	//allocate memory of host's grey data array
	h_grey_image = (unsigned char *)malloc(sizeof(unsigned char*)* total_pixels);

	//allocate and initialize memory on device
	hipMalloc(&d_rgb_image, sizeof(unsigned char) * total_pixels * CHANNELS);
	hipMalloc(&d_grey_image, sizeof(unsigned char) * total_pixels);
	hipMemset(d_grey_image, 0, sizeof(unsigned char) * total_pixels);
	
	//copy host rgb data array to device rgb data array
	hipMemcpy(d_rgb_image, h_rgb_image, sizeof(unsigned char) * total_pixels * CHANNELS, hipMemcpyHostToDevice);

	//define block and grid dimensions
	const dim3 dimGrid((int)ceil((cols)/16), (int)ceil((rows)/16));
	const dim3 dimBlock(16, 16);
	
	//execute cuda kernel
	colorConvertToGrey<<<dimGrid, dimBlock>>>(d_rgb_image, d_grey_image, rows, cols);

	//copy computed gray data array from device to host
	hipMemcpy(h_grey_image, d_grey_image, sizeof(unsigned char) * total_pixels, hipMemcpyDeviceToHost);

	//output the grayscale image
	outputImage(output_file, h_grey_image, rows, cols);
	hipFree(d_rgb_image);
	hipFree(d_grey_image);
	return 0;
}

//function for loading an image into rgb format unsigned char array
size_t loadImageFile(unsigned char *grey_image, const std::string &input_file, int *rows, int *cols) 
{
	cv::Mat img_data; //opencv Mat object

	//read image data into img_data Mat object
	img_data = cv::imread(input_file.c_str(), CV_LOAD_IMAGE_COLOR);
	if (img_data.empty()) 
	{
		std::cerr << "Unable to laod image file: " << input_file << std::endl;
	}
		
	*rows = img_data.rows;
	*cols = img_data.cols;

	//allocate memory for host rgb data array
	h_rgb_image = (unsigned char*) malloc(*rows * *cols * sizeof(unsigned char) * 3);
	unsigned char* rgb_image = (unsigned char*)img_data.data;

	//populate host's rgb data array
	int x = 0;
	for (x = 0; x < *rows * *cols * 3; x++)
	{
		h_rgb_image[x] = rgb_image[x];
	}
	
	size_t num_of_pixels = img_data.rows * img_data.cols;
	
	return num_of_pixels;
}

//function for writing gray data array to the image file
void outputImage(const std::string& output_file, unsigned char* grey_image, int rows, int cols)
{
	//serialize gray data array into opencv's Mat object
	cv::Mat greyData(rows, cols, CV_8UC1,(void *) grey_image);
	//write Mat object to file
	cv::imwrite(output_file.c_str(), greyData);
}
